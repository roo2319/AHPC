/*
** Code to implement a d2q9-bgk lattice boltzmann scheme.
** 'd2' inidates a 2-dimensional grid, and
** 'q9' indicates 9 velocities per grid cell.
** 'bgk' refers to the Bhatnagar-Gross-Krook collision step.
**
** The 'speeds' in each cell are numbered as follows:
**
** 6 2 5
**  \|/
** 3-0-1
**  /|\
** 7 4 8
**
** A 2D grid:
**
**           cols
**       --- --- ---
**      | D | E | F |
** rows  --- --- ---
**      | A | B | C |
**       --- --- ---
**
** 'unwrapped' in row major order to give a 1D array:
**
**  --- --- --- --- --- ---
** | A | B | C | D | E | F |
**  --- --- --- --- --- ---
**
** Grid indicies are:
**
**          ny
**          ^       cols(ii)
**          |  ----- ----- -----
**          | | ... | ... | etc |
**          |  ----- ----- -----
** rows(jj) | | 1,0 | 1,1 | 1,2 |
**          |  ----- ----- -----
**          | | 0,0 | 0,1 | 0,2 |
**          |  ----- ----- -----
**          ----------------------> nx
**
** Note the names of the input parameter and obstacle files
** are passed on the command line, e.g.:
**
**   ./d2q9-bgk input.params obstacles.dat
**
** Be sure to adjust the grid dimensions in the parameter file
** if you choose a different obstacle file.
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <sys/time.h>
#include <sys/resource.h>

#ifdef __APPLE__
#include <OpenCL/opencl.h>
#else
#include <hip/hip_runtime.h>
#endif

#define INDEX(ii,jj,sp,nx,ny) ((ii)+(jj)*(nx)+(sp)*(nx)*(ny))

#define NSPEEDS         9
#define FINALSTATEFILE  "final_state.dat"
#define AVVELSFILE      "av_vels.dat"

/* struct to hold the parameter values */
typedef struct
{
  int    nx;            /* no. of cells in x-direction */
  int    ny;            /* no. of cells in y-direction */
  int    maxIters;      /* no. of iterations */
  int    reynolds_dim;  /* dimension for Reynolds number */
  float density;       /* density per link */
  float accel;         /* density redistribution */
  float omega;         /* relaxation parameter */
  float w1;
  float w2;
  int free_cells;

  int reduction_count;
  int reduction_cap;

  unsigned int nworkgroupsX;
  unsigned int nworkgroupsY;
  unsigned int localnx;
  unsigned int localny;
} t_param;

/* struct to hold OpenCL objects */
typedef struct
{
  float* cells;
  float* tmp_cells;
  int* obstacles;
  float* partial_sums;
  float* averages;
} t_cuda;

__global__
void reduction(float* partial_sum,
  float* averages,
  int nGroups,
  int freeCells)
{
  int id = blockIdx.x *blockDim.x + threadIdx.x;
  float total = 0;
  for (int i = 0; i < nGroups; i++){
  total += partial_sum[i + (nGroups * id)];
  }
  averages[id] = total/(float) freeCells;
}


__global__
void accelerate_flow(float* cells,
  int* obstacles,
  int nx, int ny,
  float w1, float w2)
{
/* compute weighting factors */

/* modify the 2nd row of the grid */
int jj = ny - 2;

/* get column index */
int ii =  blockIdx.x *blockDim.x + threadIdx.x;

/* if the cell is not occupied and
** we don't send a negative density */
// 367 can be private
bool mask = (!obstacles[ii + jj* nx]
&& (cells[INDEX(ii,jj,3,nx,ny)] - w1) > 0.f
&& (cells[INDEX(ii,jj,6,nx,ny)] - w2) > 0.f
&& (cells[INDEX(ii,jj,7,nx,ny)] - w2) > 0.f);
/* increase 'east-side' densities */
cells[INDEX(ii,jj,1,nx,ny)] = mask * w1 + cells[INDEX(ii,jj,1,nx,ny)];
cells[INDEX(ii,jj,5,nx,ny)] = mask * w2 + cells[INDEX(ii,jj,5,nx,ny)];
cells[INDEX(ii,jj,8,nx,ny)] = mask * w2 + cells[INDEX(ii,jj,8,nx,ny)];
/* decrease 'west-side' densities */
cells[INDEX(ii,jj,3,nx,ny)] = mask * -w1 + cells[INDEX(ii,jj,3,nx,ny)];
cells[INDEX(ii,jj,6,nx,ny)] = mask * -w2 + cells[INDEX(ii,jj,6,nx,ny)];
cells[INDEX(ii,jj,7,nx,ny)] = mask * -w2 + cells[INDEX(ii,jj,7,nx,ny)];
}

__global__
void lbm(float* cells,
  float* tmp_cells,
  int* obstacles,
  float* partial_sum, //stores per workgroup
  int globalnx, int globalny, int localnx, int localny, 
  float omega, int iter)
  {
  extern __shared__ float local_sum[]; //stores per thread, maybe not needed
  float c_sq = 1.f / 3.f; /* square of speed of sound */
  float w0 = 4.f / 9.f;  /* weighting factor */
  float w1 = 1.f / 9.f;  /* weighting factor */
  float w2 = 1.f / 36.f; /* weighting factor */
  float tot_u = 0;          /* accumulated magnitudes of velocity for each cell */
  float speed0,speed1,speed2,speed3,speed4,speed5,speed6,speed7,speed8;
  /* get column and row indices */
  int ii = blockIdx.x * blockDim.x + threadIdx.x;
  int jj = blockIdx.y * blockDim.y + threadIdx.y;
  int lx = threadIdx.x;
  int ly = threadIdx.y;

  int idx = ii/localnx + (globalnx/localnx) * (jj/localny);
  int offset = iter * (globalnx/localnx)* (globalny/localny);

  //Array to lookup write direction
  int indexLookup[9][2] = {{0,0},{3,1},{4,2},{1,3},{2,4},{7,5},{8,6},{5,7},{6,8}};

  int y_n = (jj + 1) % globalny;
  int x_e = (ii + 1) % globalnx;
  int y_s = (jj == 0) ? (jj + globalny - 1) : (jj - 1);
  int x_w = (ii == 0) ? (ii + globalnx - 1) : (ii - 1);
  
  speed0 = cells[INDEX(ii,jj,0,globalnx,globalny)]; /* central cell, no movement */
  speed1 = cells[INDEX(x_w,jj,1,globalnx,globalny)]; /* west */
  speed2 = cells[INDEX(ii,y_s,2,globalnx,globalny)]; /* south */
  speed3 = cells[INDEX(x_e,jj,3,globalnx,globalny)]; /* east */
  speed4 = cells[INDEX(ii,y_n,4,globalnx,globalny)]; /* north */
  speed5 = cells[INDEX(x_w,y_s,5,globalnx,globalny)]; /* south-west */
  speed6 = cells[INDEX(x_e,y_s,6,globalnx,globalny)];  /* south-east */
  speed7 = cells[INDEX(x_e,y_n,7,globalnx,globalny)]; /* north-east */
  speed8 = cells[INDEX(x_w,y_n,8,globalnx,globalny)]; /* north-west */
  
  /* compute local density total */
  float local_density = 0.f;
  
  
  local_density = speed0 + speed1 + speed2 + speed3 + speed4 + speed5 + speed6 + speed7 + speed8;


  /* compute x velocity component */
  float u_x = (speed1
  + speed5
  + speed8
  - speed3
  - speed6
  - speed7)
  / local_density;
  /* compute y velocity component */
  float u_y = (speed2
  + speed5
  + speed6
  - speed4
  - speed7
  - speed8)
  / local_density;

  /* velocity squared */
  float u_sq = u_x * u_x + u_y * u_y;

  /* directional velocity components */
  float u[NSPEEDS];
  u[1] =   u_x;        /* east */
  u[2] =         u_y;  /* north */
  u[3] = - u_x;        /* west */
  u[4] =       - u_y;  /* south */
  u[5] =   u_x + u_y;  /* north-east */
  u[6] = - u_x + u_y;  /* north-west */
  u[7] = - u_x - u_y;  /* south-west */
  u[8] =   u_x - u_y;  /* south-east */

  /* equilibrium densities */
  float d_equ[NSPEEDS];
  /* zero velocity density: weight w0 */
  d_equ[0] = w0 * local_density
  * (1.f - u_sq / (2.f * c_sq));
  /* axis speeds: weight w1 */
  d_equ[1] = w1 * local_density * (1.f + u[1] / c_sq
                      + (u[1] * u[1]) / (2.f * c_sq * c_sq)
                      - u_sq / (2.f * c_sq));
  d_equ[2] = w1 * local_density * (1.f + u[2] / c_sq
                      + (u[2] * u[2]) / (2.f * c_sq * c_sq)
                      - u_sq / (2.f * c_sq));
  d_equ[3] = w1 * local_density * (1.f + u[3] / c_sq
                      + (u[3] * u[3]) / (2.f * c_sq * c_sq)
                      - u_sq / (2.f * c_sq));
  d_equ[4] = w1 * local_density * (1.f + u[4] / c_sq
                      + (u[4] * u[4]) / (2.f * c_sq * c_sq)
                      - u_sq / (2.f * c_sq));
  /* diagonal speeds: weight w2 */
  d_equ[5] = w2 * local_density * (1.f + u[5] / c_sq
                      + (u[5] * u[5]) / (2.f * c_sq * c_sq)
                      - u_sq / (2.f * c_sq));
  d_equ[6] = w2 * local_density * (1.f + u[6] / c_sq
                      + (u[6] * u[6]) / (2.f * c_sq * c_sq)
                      - u_sq / (2.f * c_sq));
  d_equ[7] = w2 * local_density * (1.f + u[7] / c_sq
                      + (u[7] * u[7]) / (2.f * c_sq * c_sq)
                      - u_sq / (2.f * c_sq));
  d_equ[8] = w2 * local_density * (1.f + u[8] / c_sq
                      + (u[8] * u[8]) / (2.f * c_sq * c_sq)
                      - u_sq / (2.f * c_sq));

  /* relaxation step */
  bool mask = !obstacles[ii+jj*globalnx];
  tmp_cells[INDEX(ii,jj,indexLookup[0][mask],globalnx,globalny)] = speed0 + mask * (omega * (d_equ[0] - speed0));
  tmp_cells[INDEX(ii,jj,indexLookup[1][mask],globalnx,globalny)] = speed1 + mask * (omega * (d_equ[1] - speed1));
  tmp_cells[INDEX(ii,jj,indexLookup[2][mask],globalnx,globalny)] = speed2 + mask * (omega * (d_equ[2] - speed2));
  tmp_cells[INDEX(ii,jj,indexLookup[3][mask],globalnx,globalny)] = speed3 + mask * (omega * (d_equ[3] - speed3));
  tmp_cells[INDEX(ii,jj,indexLookup[4][mask],globalnx,globalny)] = speed4 + mask * (omega * (d_equ[4] - speed4));
  tmp_cells[INDEX(ii,jj,indexLookup[5][mask],globalnx,globalny)] = speed5 + mask * (omega * (d_equ[5] - speed5));
  tmp_cells[INDEX(ii,jj,indexLookup[6][mask],globalnx,globalny)] = speed6 + mask * (omega * (d_equ[6] - speed6));
  tmp_cells[INDEX(ii,jj,indexLookup[7][mask],globalnx,globalny)] = speed7 + mask * (omega * (d_equ[7] - speed7));
  tmp_cells[INDEX(ii,jj,indexLookup[8][mask],globalnx,globalny)] = speed8 + mask * (omega * (d_equ[8] - speed8));


  tot_u += sqrt((u_x * u_x) + (u_y * u_y));

  // //take to outer loop
  local_sum[lx+ly*localnx] = mask?tot_u:0;

  // // Adapted from dournac.org
  for (int stride = (localnx*localny)/2; stride>0; stride /=2)
  {
  // Waiting for each 2x2 addition into given workgroup
  __syncthreads();

  // Add elements 2 by 2 between local_id and local_id + stride
  if ((lx+ly*localnx) < stride)
    local_sum[(lx+ly*localnx)] += local_sum[(lx+ly*localnx) + stride];
  }



  if (lx == 0 && ly == 0){
    partial_sum[ idx + offset ] = local_sum[0];
  }
}



/* struct to hold the 'speed' values */

/*
** function prototypes
*/

/* load params, allocate memory, load obstacles & initialise fluid particle densities */
int initialise(const char* paramfile, const char* obstaclefile,
               t_param* params, float** cells_ptr, float** tmp_cells_ptr, float** partial_sums_ptr,
               int** obstacles_ptr, float** av_vels_ptr, t_cuda* cuda);

/*
** The main calculation methods.
** timestep calls, in order, the functions:
** accelerate_flow(), propagate(), rebound() & collision()
*/

/* finalise, including freeing up allocated memory */
int finalise(const t_param* params, float** cells_ptr, float** tmp_cells_ptr,
             int** obstacles_ptr, float** av_vels_ptr, t_cuda cuda);

/* Sum all the densities in the grid.
** The total should remain constant from one timestep to the next. */
float total_density(const t_param params, float* cells);

/* compute average velocity */
float av_velocity(const t_param params, float* cells, int* obstacles, t_cuda cuda);

/* calculate Reynolds number */
float calc_reynolds(const t_param params, float average);
int write_values(const t_param params, float* cells, int* obstacles, float* av_vels);

/* utility functions */
void die(const char* message, const int line, const char* file);
void usage(const char* exe);

/*
** main program:
** initialise, timestep loop, finalise
*/

int main(int argc, char* argv[])
{
  char*    paramfile = NULL;    /* name of the input parameter file */
  char*    obstaclefile = NULL; /* name of a the input obstacle file */
  t_param  params;              /* struct to hold parameter values */
  t_cuda    cuda;                 /* struct to hold OpenCL objects */
  float* cells     = NULL;    /* grid containing fluid densities */
  float* tmp_cells = NULL;    /* scratch space */
  float* partial_sums = NULL;
  int*     obstacles = NULL;    /* grid indicating which cells are blocked */
  int free_cells = 0;
  float* av_vels   = NULL;     /* a record of the av. velocity computed for each timestep */
  struct timeval timstr;        /* structure to hold elapsed time */
  struct rusage ru;             /* structure to hold CPU time--system and user */
  double tic, toc;              /* floating point numbers to calculate elapsed wallclock time */
  double usrtim;                /* floating point number to record elapsed user CPU time */
  double systim;                /* floating point number to record elapsed system CPU time */

  /* parse the command line */
  if (argc != 3)
  {
    usage(argv[0]);
  }
  else
  {
    paramfile = argv[1];
    obstaclefile = argv[2];
  }

  /* initialise our data structures and load values from file */
  initialise(paramfile, obstaclefile, &params, &cells, &tmp_cells, &partial_sums, &obstacles, &av_vels, &cuda);

  /* iterate for maxIters timesteps */
  gettimeofday(&timstr, NULL);
  tic = timstr.tv_sec + (timstr.tv_usec / 1000000.0);

  for (int i = 0; i<params.nx * params.ny; i++){
    if (!obstacles[i]) free_cells++;
  }
  params.free_cells = free_cells;


  // Write cells and obstacles to device
  hipMemcpy(cuda.cells,cells,params.nx * params.ny * NSPEEDS * sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(cuda.obstacles,obstacles,params.nx * params.ny * sizeof(int),hipMemcpyHostToDevice);


  float* cellPointers[2] = {cuda.cells, cuda.tmp_cells};
  int read = 0;
  int write = 1;
  params.w1 = (params.density*params.accel)/9.0f;
  params.w2 = (params.density*params.accel)/36.0f;
  float* av_ptr = av_vels;
  dim3 gridsize(params.nworkgroupsX,params.nworkgroupsY);
  dim3 blocksize(params.localnx,params.localny);

  for (int tt = 0; tt < params.maxIters; tt++)
  {

    accelerate_flow<<<params.nx,1>>>(cellPointers[read], cuda.obstacles, params.nx,params.ny,params.w1,params.w2);
    
    lbm<<<gridsize,blocksize,sizeof(float) * params.localnx * params.localny>>>(cellPointers[read], cellPointers[write],cuda.obstacles,cuda.partial_sums,params.nx,params.ny,params.localnx,params.localny,params.omega,params.reduction_count);

    params.reduction_count++;
    if (params.reduction_count == params.reduction_cap){
      reduction<<<params.reduction_count,1>>>(cuda.partial_sums,cuda.averages,params.nworkgroupsX*params.nworkgroupsY,params.free_cells);
      hipMemcpy(av_ptr,cuda.averages,params.reduction_count*sizeof(float),hipMemcpyDeviceToHost);
      params.reduction_count = 0;
      av_ptr = &av_ptr[params.reduction_cap];
    }

    read ^=1; 
    write ^=1;

#ifdef DEBUG
    printf("==timestep: %d==\n", tt);
    printf("av velocity: %.12E\n", av_vels[tt]);
    printf("tot density: %.12E\n", total_density(params, cells));
#endif
  }

  // Final reduction
  if (params.reduction_count != 0){
    reduction<<<params.reduction_count,1>>>(cuda.partial_sums,cuda.averages,params.nworkgroupsX*params.nworkgroupsY,params.free_cells);
    hipMemcpy(av_ptr,cuda.averages,params.reduction_count*sizeof(float),hipMemcpyDeviceToHost);    
  }

  hipMemcpy(cells,cuda.cells,params.nx * params.ny * NSPEEDS * sizeof(float),hipMemcpyDeviceToHost);

  gettimeofday(&timstr, NULL);
  toc = timstr.tv_sec + (timstr.tv_usec / 1000000.0);
  getrusage(RUSAGE_SELF, &ru);
  timstr = ru.ru_utime;
  usrtim = timstr.tv_sec + (timstr.tv_usec / 1000000.0);
  timstr = ru.ru_stime;
  systim = timstr.tv_sec + (timstr.tv_usec / 1000000.0);

  /* write final values and free memory */
  printf("==done==\n");
  printf("Reynolds number:\t\t%.12E\n", calc_reynolds(params, av_vels[params.maxIters-1]));
  printf("Elapsed time:\t\t\t%.6lf (s)\n", toc - tic);
  printf("Elapsed user CPU time:\t\t%.6lf (s)\n", usrtim);
  printf("Elapsed system CPU time:\t%.6lf (s)\n", systim);
  write_values(params, cells, obstacles, av_vels);
  finalise(&params, &cells, &tmp_cells, &obstacles, &av_vels, cuda);

  return EXIT_SUCCESS;
}




float av_velocity(const t_param params, float* cells, int* obstacles, t_cuda cuda)
{
  int    tot_cells = 0;  /* no. of cells used in calculation */
  float tot_u;          /* accumulated magnitudes of velocity for each cell */

  /* initialise */
  tot_u = 0.f;

  /* loop over all non-blocked cells */
  for (int jj = 0; jj < params.ny; jj++)
  {
    for (int ii = 0; ii < params.nx; ii++)
    {
      /* ignore occupied cells */
      if (!obstacles[ii + jj*params.nx])
      {
        /* local density total */
        float local_density = 0.f;
        for (int kk=0; kk < NSPEEDS; kk++){
           local_density += cells[INDEX(ii,jj,kk,params.nx,params.ny)];
        }

        /* x-component of velocity */
        float u_x =  (cells[INDEX(ii,jj,1,params.nx,params.ny)]
                    + cells[INDEX(ii,jj,5,params.nx,params.ny)]
                    + cells[INDEX(ii,jj,8,params.nx,params.ny)]
                    - cells[INDEX(ii,jj,3,params.nx,params.ny)]
                    - cells[INDEX(ii,jj,6,params.nx,params.ny)]
                    - cells[INDEX(ii,jj,7,params.nx,params.ny)])
                    / local_density;
        /* compute y velocity component */
        float u_y =  (cells[INDEX(ii,jj,2,params.nx,params.ny)]
                    + cells[INDEX(ii,jj,5,params.nx,params.ny)]
                    + cells[INDEX(ii,jj,6,params.nx,params.ny)]
                    - cells[INDEX(ii,jj,4,params.nx,params.ny)]
                    - cells[INDEX(ii,jj,7,params.nx,params.ny)]
                    - cells[INDEX(ii,jj,8,params.nx,params.ny)])
                    / local_density;
        /* accumulate the norm of x- and y- velocity components */
        tot_u += sqrtf((u_x * u_x) + (u_y * u_y));
        /* increase counter of inspected cells */
        ++tot_cells;
      }
    }
  }

  return tot_u / (float)tot_cells;
}

int initialise(const char* paramfile, const char* obstaclefile,
               t_param* params, float** cells_ptr, float** tmp_cells_ptr, float** partial_sums_ptr,
               int** obstacles_ptr, float** av_vels_ptr, t_cuda *cuda)
{
  char   message[1024];  /* message buffer */
  FILE*   fp;            /* file pointer */
  int    xx, yy;         /* generic array indices */
  int    blocked;        /* indicates whether a cell is blocked by an obstacle */
  int    retval;         /* to hold return value for checking */

  /* open the parameter file */
  fp = fopen(paramfile, "r");

  if (fp == NULL)
  {
    sprintf(message, "could not open input parameter file: %s", paramfile);
    die(message, __LINE__, __FILE__);
  }

  /* read in the parameter values */
  retval = fscanf(fp, "%d\n", &(params->nx));

  if (retval != 1) die("could not read param file: nx", __LINE__, __FILE__);

  retval = fscanf(fp, "%d\n", &(params->ny));

  if (retval != 1) die("could not read param file: ny", __LINE__, __FILE__);

  retval = fscanf(fp, "%d\n", &(params->maxIters));

  if (retval != 1) die("could not read param file: maxIters", __LINE__, __FILE__);

  retval = fscanf(fp, "%d\n", &(params->reynolds_dim));

  if (retval != 1) die("could not read param file: reynolds_dim", __LINE__, __FILE__);

  retval = fscanf(fp, "%f\n", &(params->density));

  if (retval != 1) die("could not read param file: density", __LINE__, __FILE__);

  retval = fscanf(fp, "%f\n", &(params->accel));

  if (retval != 1) die("could not read param file: accel", __LINE__, __FILE__);

  retval = fscanf(fp, "%f\n", &(params->omega));

  if (retval != 1) die("could not read param file: omega", __LINE__, __FILE__);

  /* and close up the file */
  fclose(fp);

  // Local size
  params->localnx = 32;
  params->localny = 1;
  params->nworkgroupsX = params->nx/params->localnx;
  params->nworkgroupsY = params->ny/params->localny; 
  params->reduction_count = 0;
  params->reduction_cap = 10000; 

  /*
  ** Allocate memory.
  **
  ** Remember C is pass-by-value, so we need to
  ** pass pointers into the initialise function.
  **
  ** NB we are allocating a 1D array, so that the
  ** memory will be contiguous.  We still want to
  ** index this memory as if it were a (row major
  ** ordered) 2D array, however.  We will perform
  ** some arithmetic using the row and column
  ** coordinates, inside the square brackets, when
  ** we want to access elements of this array.
  **
  ** Note also that we are using a structure to
  ** hold an array of 'speeds'.  We will allocate
  ** a 1D array of these structs.
  */

  /* main grid */
 *cells_ptr = (float*) malloc(sizeof(float) * params->nx * params->ny * NSPEEDS);

  if (*cells_ptr == NULL) die("cannot allocate memory for cells", __LINE__, __FILE__);

  /* 'helper' grid, used as scratch space */
 *tmp_cells_ptr = (float*) malloc(sizeof(float) * params->nx * params->ny * NSPEEDS);
  
  if (*tmp_cells_ptr == NULL) die("cannot allocate memory for tmp_cells", __LINE__, __FILE__);

  *partial_sums_ptr = (float*) malloc(sizeof(float) * params->nworkgroupsX * params->nworkgroupsY * params->reduction_cap);

  if (*partial_sums_ptr == NULL) die("cannot allocate memory for partial_sums", __LINE__, __FILE__);

  /* the map of obstacles */
  *obstacles_ptr = (int*) malloc(sizeof(int) * (params->ny * params->nx));

  if (*obstacles_ptr == NULL) die("cannot allocate column memory for obstacles", __LINE__, __FILE__);

  /* initialise densities */
  float w0 = params->density * 4.f / 9.f;
  float w1 = params->density      / 9.f;
  float w2 = params->density      / 36.f;

  for (int jj = 0; jj < params->ny; jj++)
  {
    for (int ii = 0; ii < params->nx; ii++)
    {
      /* centre */
      (*cells_ptr)[INDEX(ii,jj,0,params->nx,params->ny)] = w0;
      /* axis directions */
      (*cells_ptr)[INDEX(ii,jj,1,params->nx,params->ny)] = w1;
      (*cells_ptr)[INDEX(ii,jj,2,params->nx,params->ny)] = w1;
      (*cells_ptr)[INDEX(ii,jj,3,params->nx,params->ny)] = w1;
      (*cells_ptr)[INDEX(ii,jj,4,params->nx,params->ny)] = w1;
      /* diagonals */
      (*cells_ptr)[INDEX(ii,jj,5,params->nx,params->ny)] = w2;
      (*cells_ptr)[INDEX(ii,jj,6,params->nx,params->ny)] = w2;
      (*cells_ptr)[INDEX(ii,jj,7,params->nx,params->ny)] = w2;
      (*cells_ptr)[INDEX(ii,jj,8,params->nx,params->ny)] = w2;
    }
  }

  /* first set all cells in obstacle array to zero */
  for (int jj = 0; jj < params->ny; jj++)
  {
    for (int ii = 0; ii < params->nx; ii++)
    {
      (*obstacles_ptr)[ii + jj*params->nx] = 0;
    }
  }

  /* open the obstacle data file */
  fp = fopen(obstaclefile, "r");

  if (fp == NULL)
  {
    sprintf(message, "could not open input obstacles file: %s", obstaclefile);
    die(message, __LINE__, __FILE__);
  }

  /* read-in the blocked cells list */
  while ((retval = fscanf(fp, "%d %d %d\n", &xx, &yy, &blocked)) != EOF)
  {
    /* some checks */
    if (retval != 3) die("expected 3 values per line in obstacle file", __LINE__, __FILE__);

    if (xx < 0 || xx > params->nx - 1) die("obstacle x-coord out of range", __LINE__, __FILE__);

    if (yy < 0 || yy > params->ny - 1) die("obstacle y-coord out of range", __LINE__, __FILE__);

    if (blocked != 1) die("obstacle blocked value should be 1", __LINE__, __FILE__);

    /* assign to array */
    (*obstacles_ptr)[xx + yy*params->nx] = blocked;
  }

  /* and close the file */
  fclose(fp);

  /*
  ** allocate space to hold a record of the avarage velocities computed
  ** at each timestep
  */
  *av_vels_ptr = (float*)malloc(sizeof(float) * params->maxIters);


  // Allocate memory on device
  hipMalloc((void **)&cuda->cells, params->nx * params->ny * NSPEEDS * sizeof(float));
  hipMalloc((void **)&cuda->tmp_cells, params->nx * params->ny * NSPEEDS * sizeof(float));
  hipMalloc((void **)&cuda->obstacles, params->nx * params->ny * sizeof(int));
  hipMalloc((void **)&cuda->partial_sums, params->nworkgroupsX * params->nworkgroupsY * params->reduction_cap * sizeof(float));
  hipMalloc((void **)&cuda->averages, params->reduction_cap * sizeof(float));
  

  return EXIT_SUCCESS;
}

int finalise(const t_param* params, float** cells_ptr, float** tmp_cells_ptr,
             int** obstacles_ptr, float** av_vels_ptr, t_cuda cuda)
{
  /*
  ** free up allocated memory
  */
  free(*cells_ptr);
  *cells_ptr = NULL;

  free(*tmp_cells_ptr);
  *tmp_cells_ptr = NULL;

  free(*obstacles_ptr);
  *obstacles_ptr = NULL;

  free(*av_vels_ptr);
  *av_vels_ptr = NULL;


  hipFree(cuda.cells);
  hipFree(cuda.tmp_cells);
  hipFree(cuda.obstacles);

  return EXIT_SUCCESS;
}


float calc_reynolds(const t_param params, float average)
{
  const float viscosity = 1.f / 6.f * (2.f / params.omega - 1.f);

  return average * params.reynolds_dim / viscosity;
}

float total_density(const t_param params, float* cells)
{
  float total = 0.f;  /* accumulator */

  for (int jj = 0; jj < params.ny; jj++)
  {
    for (int ii = 0; ii < params.nx; ii++)
    {
      total = 0;
      for (int kk=0; kk < NSPEEDS; kk++){
        total += cells[INDEX(ii,jj,kk,params.nx,params.ny)];
      }
    }
  }

  return total;
}

int write_values(const t_param params, float* cells, int* obstacles, float* av_vels)
{
  FILE* fp;                     /* file pointer */
  const float c_sq = 1.f / 3.f; /* sq. of speed of sound */
  float local_density;         /* per grid cell sum of densities */
  float pressure;              /* fluid pressure in grid cell */
  float u_x;                   /* x-component of velocity in grid cell */
  float u_y;                   /* y-component of velocity in grid cell */
  float u;                     /* norm--root of summed squares--of u_x and u_y */

  fp = fopen(FINALSTATEFILE, "w");

  if (fp == NULL)
  {
    die("could not open file output file", __LINE__, __FILE__);
  }

  for (int jj = 0; jj < params.ny; jj++)
  {
    for (int ii = 0; ii < params.nx; ii++)
    {
      /* an occupied cell */
      if (obstacles[ii + jj*params.nx])
      {
        u_x = u_y = u = 0.f;
        pressure = params.density * c_sq;
      }
      /* no obstacle */
      else
      {
        local_density = 0;
        for (int kk=0; kk < NSPEEDS; kk++){
          local_density += cells[INDEX(ii,jj,kk,params.nx,params.ny)];
        }

        /* compute x velocity component */
        float u_x =  (cells[INDEX(ii,jj,1,params.nx,params.ny)]
                    + cells[INDEX(ii,jj,5,params.nx,params.ny)]
                    + cells[INDEX(ii,jj,8,params.nx,params.ny)]
                    - cells[INDEX(ii,jj,3,params.nx,params.ny)]
                    - cells[INDEX(ii,jj,6,params.nx,params.ny)]
                    - cells[INDEX(ii,jj,7,params.nx,params.ny)])
                    / local_density;
        /* compute y velocity component */
        float u_y =  (cells[INDEX(ii,jj,2,params.nx,params.ny)]
                    + cells[INDEX(ii,jj,5,params.nx,params.ny)]
                    + cells[INDEX(ii,jj,6,params.nx,params.ny)]
                    - cells[INDEX(ii,jj,4,params.nx,params.ny)]
                    - cells[INDEX(ii,jj,7,params.nx,params.ny)]
                    - cells[INDEX(ii,jj,8,params.nx,params.ny)])
                    / local_density;
        /* compute norm of velocity */
        u = sqrtf((u_x * u_x) + (u_y * u_y));
        /* compute pressure */
        pressure = local_density * c_sq;
      }

      /* write to file */
      fprintf(fp, "%d %d %.12E %.12E %.12E %.12E %d\n", ii, jj, u_x, u_y, u, pressure, obstacles[ii * params.nx + jj]);
    }
  }

  fclose(fp);

  fp = fopen(AVVELSFILE, "w");

  if (fp == NULL)
  {
    die("could not open file output file", __LINE__, __FILE__);
  }

  for (int ii = 0; ii < params.maxIters; ii++)
  {
    fprintf(fp, "%d:\t%.12E\n", ii, av_vels[ii]);
  }

  fclose(fp);

  return EXIT_SUCCESS;
}


void die(const char* message, const int line, const char* file)
{
  fprintf(stderr, "Error at line %d of file %s:\n", line, file);
  fprintf(stderr, "%s\n", message);
  fflush(stderr);
  exit(EXIT_FAILURE);
}

void usage(const char* exe)
{
  fprintf(stderr, "Usage: %s <paramfile> <obstaclefile>\n", exe);
  exit(EXIT_FAILURE);
}

#define MAX_DEVICES 32
#define MAX_DEVICE_NAME 1024

